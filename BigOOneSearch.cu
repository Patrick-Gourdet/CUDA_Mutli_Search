#include "hip/hip_runtime.h"

// ReSharper disable CppClangTidyCppcoreguidelinesAvoidGoto
// ReSharper disable CppClangTidyHicppAvoidGoto
#include <cassert>
#include <iostream>
#include <limits>
#include "hip/hip_runtime.h"
#include ""

struct SetSearchValue
{
private:
	int search = 0;
public:
	 SetSearchValue(int serarch_in) : search(serarch_in){}
	int* getSearch(){return &search;};
	
};
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
  	goto ERROR;
  }
  return result;
}


__global__ void search_bigo_one(int* d_in,int* d_out_result, const int* d_in_search_val,const int SIZE)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < SIZE){
	const int value = d_in[index];	
		if(value == *d_in_search_val)
		{
			printf("%d\n", value);
			*d_out_result = blockIdx.x * blockDim.x + threadIdx.x;		
		}
	}
}
/// <summary>
/// This is a short intro to a GPU search algorithm
/// this in theory should have run time of O(1)
/// the algorithm assigns one thread to each element in the array.
/// and the element that we are searching for is passed in along with
/// the thread and array element then the thread that contains the
/// element returns the 
/// </summary>
/// <returns></returns>
auto main() -> int
{
	const auto SIZE = 1000;
	const auto BYTES = SIZE * sizeof(int);
	auto* h_in = static_cast<int*>(malloc(SIZE * sizeof(int)));
	
	int input = 0;
	do
	{
		std::cout << "Enter value to search for: " << std::endl;
		std::cin >> input;
	}while (input == 0);

	SetSearchValue sv =  SetSearchValue(input);
	auto max = std::numeric_limits<int>::max();
	int* d_in;
	int* d_result_index;
	int* d_search;
	auto initializer = 0;
	auto* const search = sv.getSearch();
	auto* const index = &max;
	h_in[544] = *search;
	
	for(int i{0} ; i < SIZE; i++)
	{
		if(i == 52) continue;
		 initializer = rand() % 1472 + 1;
		*(h_in + i)  = initializer == 44? initializer * initializer : initializer;
		std::cout << *(h_in + i);
	}

	for(int ii = 0 ; ii < SIZE ; ii++)
	{
		std::cout << ":   this is the input value at" << *(h_in + ii) << std::endl;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	checkCuda(hipMalloc(reinterpret_cast<void**>(&d_in), BYTES));
	checkCuda(hipMalloc(reinterpret_cast<void**>(&d_search), sizeof(int)));
	checkCuda(hipMalloc(reinterpret_cast<void**>(&d_result_index), sizeof(int)));

	checkCuda(hipMemcpy(d_in, h_in, BYTES, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(d_search, search, sizeof(int), hipMemcpyHostToDevice));
	const auto threads = 256;
	auto block = (SIZE * threads)/ threads;
	search_bigo_one<<<block,threads>>>(d_in,d_result_index,d_search,SIZE);
	checkCuda(hipGetLastError());
	checkCuda(hipMemcpy(index, d_result_index, sizeof(int), hipMemcpyDeviceToHost));
	
	if(*index == max)
	std::cout << "The value was found at: "  << *index << std::endl;
	else
	std::cout << "No such value" << std::endl;

ERROR:
    checkCuda(hipFree(d_in));
	checkCuda(hipFree(d_search));
	checkCuda(hipFree(d_result_index));
	free(index);
	free(h_in);


}

